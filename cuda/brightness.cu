#include "hip/hip_runtime.h"

__device__ int clamp(int value, int low, int high) {
    return fmaxf(low, fminf(value, high));
}

__global__ void brightnessKernel(const uchar *input, uchar *output, int rows, int cols, int bright) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < rows && x < cols) {
        int idx = (y * cols + x) * 3;
        output[idx]= (uchar)clamp(input[idx]+(float)bright,0.0f,255.0f); // Blue
        output[idx + 1] = (uchar)clamp(input[idx+1]+(float)bright,0.0f,255.0f); // Green
        output[idx + 2] = (uchar)clamp(input[idx+2]+(float)bright,0.0f,255.0f); // Red
    }
}

void ParallelBrightnessCUDA(uchar *input,uchar *output, int rows, int cols, int bright) {

    // Input and output data
    size_t dataSize = rows * cols * 3 * sizeof(uchar);
    uchar *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);

    // Copy input data to device
    hipMemcpy(d_input, input, dataSize, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    brightnessKernel<<<gridSize, blockSize>>>(d_input, d_output, rows, cols, bright);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy output data back to host
    hipMemcpy(output, d_output, dataSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}
