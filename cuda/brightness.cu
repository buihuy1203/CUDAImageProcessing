
#include <hip/hip_runtime.h>
static float BrightTime=0.0f;

static __device__ int clamp(int value, int low, int high) {
    return fmaxf(low, fminf(value, high));
}

__global__ void brightnessKernel(const unsigned char *input, unsigned char *output, int rows, int cols, int bright) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < rows && x < cols) {
        int idx = (y * cols + x) * 3;
        output[idx]= (unsigned char)clamp(input[idx]+(float)bright,0.0f,255.0f); // Blue
        output[idx + 1] = (unsigned char)clamp(input[idx+1]+(float)bright,0.0f,255.0f); // Green
        output[idx + 2] = (unsigned char)clamp(input[idx+2]+(float)bright,0.0f,255.0f); // Red
    }
}

__host__ void ParallelBrightnessCUDA(unsigned char *input,unsigned char *output, int rows, int cols, int bright) {

    // Input and output data
    size_t dataSize = rows * cols * 3 * sizeof(unsigned char);
    unsigned char *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);

    // Create Event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy input data to device
    hipMemcpy(d_input, input, dataSize, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);

    // Record start time
    hipEventRecord(start); 

    // Launch kernel
    brightnessKernel<<<gridSize, blockSize>>>(d_input, d_output, rows, cols, bright);

    // Record stop time
    hipEventRecord(stop);

    // Wait for GPU to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    BrightTime += milliseconds;

    // Copy output data back to host
    hipMemcpy(output, d_output, dataSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

__host__ float getBrightTime() {
    return BrightTime;
}