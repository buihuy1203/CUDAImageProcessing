
#include <hip/hip_runtime.h>
#define M_PI 3.14159
static __device__ int clamp(int value, int low, int high) {
    return max(low, min(value, high));
}

__global__ void blurKernel(unsigned char *input, unsigned char *output,float *kernel1D,  int rows, int cols, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int halfSize = kernelSize / 2;

    if (x>=0 && x < cols && y < rows && y>=0) {
        float sum[3] = {0.0f, 0.0f, 0.0f};

        for (int kx = -halfSize; kx <= halfSize; ++kx) {
            for (int ky = -halfSize; ky <= halfSize; ++ky) {
                int px = clamp(x + kx, 0, cols - 1);
                int py = clamp(y + ky, 0, rows - 1);

                int pixelIndex = (py * cols + px) * 3;
                float weight = kernel1D[(kx + halfSize) * kernelSize + (ky + halfSize)];

                sum[0] += (float)input[pixelIndex] * weight;       // Blue channel
                sum[1] += (float)input[pixelIndex + 1] * weight;   // Green channel
                sum[2] += (float)input[pixelIndex + 2] * weight;   // Red channel
            }
        }

        int outputIndex = (y * cols + x) * 3;
        output[outputIndex] = (unsigned char)clamp(sum[0], 0.0f, 255.0f);
        output[outputIndex + 1] = (unsigned char)clamp(sum[1], 0.0f, 255.0f);
        output[outputIndex + 2] = (unsigned char)clamp(sum[2], 0.0f, 255.0f);
    }
}

__host__ float* createGaussianKernel(int size, float sigma) {
    float* kernel1D = new float[size * size]; // Cấp phát bộ nhớ cho mảng 1D
    float sum = 0.0f;
    int halfSize = size / 2;

    // Tạo kernel Gaussian
    for (int x = -halfSize; x <= halfSize; ++x) {
        for (int y = -halfSize; y <= halfSize; ++y) {
            int idx = (x + halfSize) * size + (y + halfSize); // Biến đổi chỉ số 2D thành 1D
            kernel1D[idx] = (1.0f / (2.0f * M_PI * sigma * sigma)) * expf(-(x * x + y * y) / (2.0f * sigma * sigma));
            sum += kernel1D[idx];
        }
    }

    // Chuẩn hóa kernel
    for (int i = 0; i < size * size; ++i) {
        kernel1D[i] /= sum;
    }

    return kernel1D; // Trả về con trỏ mảng
}

__host__ void ParallelBlurCUDA(unsigned char *input,unsigned char *output,int rows, int cols, float blur_sar) {
    // Gaussian kernel
    int kernelSize = 7;
    // Input and output data
    size_t dataSize = rows * cols * 3 * sizeof(unsigned char);
    unsigned char *d_input, *d_output;
    float *d_kernel;
    float *kernel1D = createGaussianKernel(kernelSize, blur_sar);
    // Allocate device memory
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);
    hipMalloc(&d_kernel, 7 * 7 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel1D, 7 * 7 * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    blurKernel<<<gridSize, blockSize>>>(d_input, d_output, d_kernel, rows, cols, kernelSize);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy output data back to host
    hipMemcpy(output, d_output, dataSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
    delete[] kernel1D;

}
