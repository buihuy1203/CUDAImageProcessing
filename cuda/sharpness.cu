
#include <hip/hip_runtime.h>
static __device__ int clamp(int value, int low, int high) {
    return fmaxf(low, fminf(value, high));
}

__global__ void sharpenKernel(const unsigned char *input, unsigned char *output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int kernel2D[3][3] = {
            {0, 1, 0},
            {1, -4, 1},
            {0, 1, 0}
        };
    if (y > 0 && y < rows - 1 && x > 0 && x < cols - 1) {
        int sum = 0;
        for (int k = -1; k <= 1; ++k) {
            for (int l = -1; l <= 1; ++l) {
                if ((y + k) >= 0 && (y + k) < rows && (x + l) >= 0 && (x + l) < cols) {
                    sum += kernel2D[k + 1][l + 1] * input[(y + k) * cols + (x + l)];
                }
            }
        }
        output[y * cols + x] = (unsigned char)clamp(sum, 0, 255);
    }
}

__global__ void applysharpenKernel(const unsigned char *input,const unsigned char *result, unsigned char *output, int rows, int cols, float sharp) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        for (int c = 0; c < 3; c++) {
            int idx = (y * cols + x) * 3 + c;
            int grayIdx = y * cols + x;
            int newValue = (float)input[idx] - sharp * (float)result[grayIdx];
            output[idx] = (unsigned char)clamp(newValue, 0, 255);
        }
    }
}

void ParallelSharpCUDA(unsigned char *inputColor, unsigned char *inputGray, unsigned char *output, int rows, int cols, float sharp_var) {

    // Input and output data
    size_t dataGray = rows * cols * sizeof(unsigned char);
    size_t dataSize = rows * cols * 3 * sizeof(unsigned char);
    unsigned char *d_inputData, *d_output, *d_inputColor, *d_result;

    // Allocate device memory
    hipMalloc(&d_inputData, dataGray);
    hipMalloc(&d_output, dataSize);
    hipMalloc(&d_inputColor, dataSize);
    hipMalloc(&d_result, dataGray);

    // Copy input data to device
    hipMemcpy(d_inputData, inputGray, dataGray, hipMemcpyHostToDevice);
    hipMemcpy(d_inputColor, inputColor, dataSize, hipMemcpyHostToDevice);
    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    sharpenKernel<<<gridSize, blockSize>>>(d_inputData, d_result, rows, cols);
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    applysharpenKernel<<<gridSize, blockSize>>>(d_inputColor, d_result, d_output, rows, cols, sharp_var);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy output data back to host
    hipMemcpy(output, d_output, dataSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_inputData);
    hipFree(d_output);
    hipFree(d_result);
    hipFree(d_inputColor);
}
